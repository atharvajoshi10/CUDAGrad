
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define MATRIX_DIM 1024
#define BLOCK_SIZE 32
#define GRID_SIZE 32

__global__ void matmul(int *a, int *b, int *c){
    int threadRow = threadIdx.x;
    int threadCol = threadIdx.y;

    int blockRow = blockIdx.x;
    int blockCol = blockIdx.y;

    int outputRow = (blockRow * BLOCK_SIZE) + threadRow;
    int outputCol = (blockCol * BLOCK_SIZE) + threadCol;
    int local_c = 0;
    __shared__ int A[BLOCK_SIZE][BLOCK_SIZE], B[BLOCK_SIZE][BLOCK_SIZE];
    
    for(int tileIndex = 0; tileIndex < GRID_SIZE; tileIndex ++){
        A[threadRow][threadCol] = a[(MATRIX_DIM * outputRow) + ((tileIndex * BLOCK_SIZE) + threadCol)];
        B[threadRow][threadCol] = b[((tileIndex * BLOCK_SIZE + threadRow) * MATRIX_DIM) + (outputCol)];
        __syncthreads();
        for (size_t i = 0; i < BLOCK_SIZE; i++)
        {
            local_c += A[threadRow][i] * B[i][threadCol];
        }
        __syncthreads();
         
    }
    c[outputRow * MATRIX_DIM + outputCol] = local_c;
}



int main(){
    int i;
    const auto size = MATRIX_DIM * MATRIX_DIM;
    int *a = (int*)malloc(sizeof(int) * size);          
    int *b = (int*)malloc(sizeof(int) * size);          
    int *c = (int*)malloc(sizeof(int) * size);

    for(i=0; i<size; i++){
        a[i]=1;
        b[i]=2;
  	}
    int *gpu_a, *gpu_b, *gpu_c;

    hipMalloc((void**)&gpu_a, sizeof(int)*size); 
    hipMalloc((void**)&gpu_b, sizeof(int)*size);
    hipMalloc((void**)&gpu_c, sizeof(int)*size);
    struct timespec start, stop; 
    double time;


    hipMemcpy(gpu_a, a, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, sizeof(int)*size, hipMemcpyHostToDevice);

    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}

    matmul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);				
    hipMemcpy(c, gpu_c, sizeof(int)*size, hipMemcpyDeviceToHost);
    
    if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
    printf("time is %f ns\n", time*1e9);

    printf("c[451][451] = %d\n", c[451 * MATRIX_DIM + 451]);
  	
    free(a);
    free(b);
    free(c);
    hipFree(gpu_a);  
    hipFree(gpu_b);  
    hipFree(gpu_c);
    return 0;
}